#include "hip/hip_runtime.h"
#include "external/book-professional-cuda-c-programming/CodeSamples/common/common.h"

#include <hip/hip_runtime.h>

#include <iostream>

#include <stdio.h>

static void info() {
	// http://developer.download.nvidia.com/compute/cuda/2_3/toolkit/docs/online/structcudaDeviceProp.html
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Total Global Memory (MBytes): %zu\n",
				prop.totalGlobalMem / 1024 / 1024);
		printf("  Warp Size: %d\n", prop.warpSize);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
}

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. A 1D thread block and 1D grid are used. sumArraysOnHost sequentially
 * iterates through vector elements on the host.
 */

void initialData(float *ip, const int size) {
	int i;

	for (i = 0; i < size; i++) {
		ip[i] = (float) (rand() & 0xFF) / 10.0f;
	}

	return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
	float *ia = A;
	float *ib = B;
	float *ic = C;

	for (int iy = 0; iy < ny; iy++) {
		for (int ix = 0; ix < nx; ix++) {
			ic[ix] = ia[ix] + ib[ix];

		}

		ia += nx;
		ib += nx;
		ic += nx;
	}

	return;
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < N; i++) {
		if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
			match = 0;
			printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
			break;
		}
	}

	if (match)
		printf("Arrays match.\n\n");
	else
		printf("Arrays do not match.\n\n");
}

// grid 1D block 1D
__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx,
		int ny) {
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

	if (ix < nx)
		for (int iy = 0; iy < ny; iy++) {
			int idx = iy * nx + ix;
			MatC[idx] = MatA[idx] + MatB[idx];
		}

}

int to_mb(int bytes) {
	return bytes / 1024 / 1024;
}

int main(int argc, char **argv) {
	printf("%s Starting...\n", argv[0]);

	info();

	size_t mem_tot_0 = 0;
	size_t mem_free_0 = 0;
//	size_t mem_tot_1 = 0;
//	size_t mem_free_1 = 0;
//	// out of mem
//	hipSetDevice(0);
//	hipDeviceSynchronize();
//	hipDeviceSynchronize();

	hipDeviceReset();
	hipMemGetInfo(&mem_free_0, &mem_tot_0);
	std::cout << "Free memory before copy dev 0: " << mem_free_0 / 1024 / 1024
			<< std::endl;

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of matrix
	int nx = 1 << 12; //14;
	int ny = 1 << 12; // 14;

	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);
	printf("Matrix size: nx %d ny %d nMBytes %d\n", nx, ny, to_mb(nBytes));

	// malloc host memory
	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *) malloc(nBytes);
	h_B = (float *) malloc(nBytes);
	hostRef = (float *) malloc(nBytes);
	gpuRef = (float *) malloc(nBytes);

	// initialize data at host side
	double iStart = seconds();
	initialData(h_A, nxy);
	initialData(h_B, nxy);
	double iElaps = seconds() - iStart;
	printf("initialize matrix elapsed %f sec\n", iElaps);

	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// add matrix at host side for result checks
	iStart = seconds();
	sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
	iElaps = seconds() - iStart;
	printf("sumMatrixOnHost elapsed %f sec\n", iElaps);

	// malloc device global memory
	float *d_MatA, *d_MatB, *d_MatC;
	CHECK(hipMalloc((void ** )&d_MatA, nBytes));
	CHECK(hipMalloc((void ** )&d_MatB, nBytes));
	CHECK(hipMalloc((void ** )&d_MatC, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int dimx = 32;
	dim3 block(dimx, 1);
	dim3 grid((nx + block.x - 1) / block.x, 1);

	iStart = seconds();
	sumMatrixOnGPU1D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
	CHECK(hipDeviceSynchronize());
	iElaps = seconds() - iStart;
	printf("sumMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
			grid.y, block.x, block.y, iElaps);

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nxy);

	// free device global memory
	CHECK(hipFree(d_MatA));
	CHECK(hipFree(d_MatB));
	CHECK(hipFree(d_MatC));

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	// reset device
	CHECK(hipDeviceReset());

	return (0);
}
